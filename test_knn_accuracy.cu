#include "hip/hip_runtime.h"
#include <cassert>
#include <cstdio>
#include <fstream>
#include <iostream>
#include <set>
#include <string>
#include <utility>
// faiss
#include "vendor/faiss/faiss/gpu/GpuIndexFlat.h"
#include "vendor/faiss/faiss/gpu/StandardGpuResources.h"
// cmdline
#include "vendor/cmdline/cmdline.h"
// qvis
#include "handle_cuda_err.hpp"
#include "qvis_io.h"

using namespace std;

__global__ void kernel_fill_labels(unsigned points_num, unsigned K, unsigned aligned_num, long *knn_indicates_device,
                                   unsigned *labels, unsigned *knn_labels) {
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    for (; idx < points_num; idx += gridDim.x * blockDim.x) {
        for (int i = 0; i < K + 1; i++) {
            knn_labels[i * aligned_num + idx] = labels[knn_indicates_device[idx * (K + 1) + i]];
        }
    }
}

__global__ void kernel_knn_label(unsigned points_num, unsigned K, unsigned aligned_num, unsigned *knn_labels) {
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned     t;
    for (; idx < points_num; idx += gridDim.x * blockDim.x) {
        // bubble sort label [1..K]
        for (int i = 1; i < K; i++) {
            for (int j = 1; j < K; j++) {
                unsigned &a = knn_labels[j * aligned_num + idx];
                unsigned &b = knn_labels[(j + 1) * aligned_num + idx];
                if (a > b) {
                    t = a;
                    a = b;
                    b = t;
                }
            }
        }
        // find common number
        int      result_times = 0, current_times = 0;
        unsigned result_label, current_label     = 0xffffffff;
        for (int i = 1; i <= K; i++) {
            if (knn_labels[i * aligned_num + idx] == current_label) {
                current_times++;

            } else {
                current_times = 1;
                current_label = knn_labels[i * aligned_num + idx];
            }
            if (current_times > result_times) {
                result_label = current_label;
                result_times = current_times;
            }
        }

        // save result
        // knn_labels[aligned_num + idx] = result_label;
        knn_labels[idx] = result_label;
    }
}
set<int> parse_ks(const string &str) {
    set<int> ks;
    int      num = 0;
    for (size_t i = 0; i < str.size(); i++) {
        if (str[i] == ' ' || str[i] == '\t') {
            continue;
        }
        if (str[i] == ',') {
            ks.insert(num);
            num = 0;
            continue;
        }
        if (str[i] >= '0' && str[i] <= '9') {
            num *= 10;
            num += str[i] - '0';
        }
    }
    if (num) {
        ks.insert(num);
    }
    return ks;
}
int main(int argc, char **argv) {
    cmdline::parser parser;
    parser.add<string>("datafile", 'd', "lowdim vector file path", true, "");
    parser.add<string>("labelfile", 'l', "label file path", true, "");
    parser.add<int>("k", 'k', "number neighborhood", false, 5);
    parser.add<string>("ks", '\0', "test multiple K at same time, Ks must seperated by comma", false, "");
    parser.parse_check(argc, argv);

    // parse Ks
    set<int> ks = parse_ks(parser.get<string>("ks"));
    if (parser.exist("k") || ks.size() == 0) {
        ks.insert(parser.get<int>("k"));
    }
    int K = *ks.rbegin();
    printf("Ks: ");
    for (auto it = ks.begin(); it != ks.end(); it++) {
        printf("%d ", *it);
    }
    printf("\n");

    //
    unsigned  points_num, labels_num, dim;
    float *   data, *data_device;
    unsigned *labels = nullptr;

    // load data
    load_data(parser.get<string>("datafile").c_str(), data, points_num, dim);
    printf("Data load successful, N = %u, dim = %u\n", points_num, dim);

    // load label
    load_label(parser.get<string>("labelfile").c_str(), labels, &labels_num);
    printf("Labels laod successful, N = %u\n", labels_num);
    assert(points_num == labels_num);

    // build knn graph
    HANDLE_ERROR(hipMallocManaged((void **)&data_device, sizeof(float) * points_num * dim));
    HANDLE_ERROR(hipMemcpy(data_device, data, sizeof(float) * points_num * dim, hipMemcpyHostToDevice));
    faiss::gpu::StandardGpuResources gpuresource;
    faiss::gpu::GpuIndexFlat *       data_index = nullptr;
    data_index                                  = new faiss::gpu::GpuIndexFlat(&gpuresource, dim, faiss::METRIC_L2);
    data_index->add(points_num, data_device);

    // search for knn
    float *knn_distances_device;
    long * knn_indicates_device;

    HANDLE_ERROR(hipMallocManaged((void **)&knn_distances_device, sizeof(float) * points_num * (K + 1)));
    HANDLE_ERROR(hipMallocManaged((void **)&knn_indicates_device, sizeof(long) * points_num * (K + 1)));
    data_index->search(points_num, data_device, K + 1, knn_distances_device, knn_indicates_device);
    HANDLE_ERROR(hipFree(knn_distances_device));
    HANDLE_ERROR(hipFree(data_device));

    // get labels
    unsigned *knn_labels, *labels_device;
    unsigned  aligned_num = (points_num + 63) / 64 * 64; // aligned to 256 bytes;

    HANDLE_ERROR(hipMallocManaged((void **)&knn_labels, sizeof(unsigned) * aligned_num * (K + 1)));
    HANDLE_ERROR(hipMallocManaged((void **)&labels_device, sizeof(unsigned) * points_num));
    HANDLE_ERROR(hipMemcpy(labels_device, labels, sizeof(float) * points_num, hipMemcpyHostToDevice));

    const int ThreadPerBlock = 256;
    kernel_fill_labels<<<50, ThreadPerBlock>>>(points_num, K, aligned_num, knn_indicates_device, labels_device,
                                               knn_labels);
    HANDLE_ERROR(hipDeviceSynchronize());
    HANDLE_ERROR(hipFree(labels_device));
    HANDLE_ERROR(hipFree(knn_indicates_device));

    // print result
    printf("K\tratio\tcorrect\tsample_number\n");
    for (auto k = ks.begin(); k != ks.end(); k++) {
        // sort labels and get most common

        kernel_knn_label<<<50, ThreadPerBlock>>>(points_num, *k, aligned_num, knn_labels);
        HANDLE_ERROR(hipDeviceSynchronize());
        int equal_num = 0;
        for (unsigned i = 0; i < points_num; i++) {
            equal_num += knn_labels[i] == labels[i];
        }
        // sum up
        printf("%d\t%f\t%d\t%u\n", *k, double(equal_num) / points_num, equal_num, points_num);
    }

    HANDLE_ERROR(hipFree(knn_labels));
    return 0;
}
